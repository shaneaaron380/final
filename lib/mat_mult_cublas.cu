#include "mat_mult_cublas.h"

/*******************************************************************************
 * 
 * Solve the following:
 * 
 *	A * X = alpha * B
 *	    X = alpha * A^(-1) * B
 * 
 ******************************************************************************/

#define N 2

int MatMultCublas(const Matrix A, Matrix B)
{
	/*float A[N][N] =   { { 3.0, -1.0 },*/
	/*                    { 0.0, -2.0 } },*/

	/*      B[N][N] =   { { 1.0, 1.0 },*/
	/*                    { 1.0, 1.0 } };*/

	if (cublasInit() != HIPBLAS_STATUS_SUCCESS)
		RET_ERROR("cublasInit failed");

	Matrix d_A, d_B;

#if 0
	printf("using static, NON-transposed matrices\n");

	float As[7][5] =  { { 3.0, -1.0,  2.0,  2.0,  1.0 },
						{ 0.0, -2.0,  4.0, -1.0,  3.0 },
						{ 0.0,  0.0, -3.0,  0.0,  2.0 },
						{ 0.0,  0.0,  0.0,  4.0, -2.0 },
						{ 0.0,  0.0,  0.0,  0.0,  1.0 },
						{ 0.0,  0.0,  0.0,  0.0,  0.0 },
						{ 0.0,  0.0,  0.0,  0.0,  0.0 } };

	float Bs[6][3] =  { {   6.0, 10.0,  -2.0 },
						{ -16.0, -1.0,   6.0 },
						{  -2.0,  1.0,  -4.0 },
						{  14.0,  0.0, -14.0 },
						{  -1.0,  2.0,   1.0 },
						{   0.0,  0.0,   0.0 } };
#elif 0
	printf("using static, transposed matrices\n");

	float As[5][7] =  { {  3.0,  0.0,  0.0,  0.0,  0.0,  0.0,  0.0 },
						{ -1.0, -2.0,  0.0,  0.0,  0.0,  0.0,  0.0 },
						{  2.0,  4.0, -3.0,  0.0,  0.0,  0.0,  0.0 },
						{  2.0, -1.0,  0.0,  4.0,  0.0,  0.0,  0.0 },
						{  1.0,  3.0,  2.0, -2.0,  1.0,  0.0,  0.0 } };

	float Bs[3][6] =  { {   6.0, -16.0,  -2.0,  14.0,  -1.0,   0.0 },
						{  10.0,  -1.0,   1.0,   0.0,   2.0,   0.0 },
						{  -2.0,   6.0,  -4.0, -14.0,   1.0,   0.0 } };

	cublasAlloc(5*7, sizeof(float), (void**) &d_A.els);
	cublasAlloc(3*6, sizeof(float), (void**) &d_B.els);

	hipMemcpy(d_A.els, As, 5*7*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B.els, Bs, 6*3*sizeof(float), hipMemcpyHostToDevice);

#else
	printf("using static, transposed matrices without padding\n");

	float As[5][5] =  { {  3.0,  0.0,  0.0,  0.0,  0.0},
						{ -1.0, -2.0,  0.0,  0.0,  0.0},
						{  2.0,  4.0, -3.0,  0.0,  0.0},
						{  2.0, -1.0,  0.0,  4.0,  0.0},
						{  1.0,  3.0,  2.0, -2.0,  1.0} };

	float Bs[3][5] =  { {   6.0, -16.0,  -2.0,  14.0,  -1.0},
						{  10.0,  -1.0,   1.0,   0.0,   2.0},
						{  -2.0,   6.0,  -4.0, -14.0,   1.0} };

	cublasAlloc(5*5, sizeof(float), (void**) &d_A.els);
	cublasAlloc(3*5, sizeof(float), (void**) &d_B.els);

	/*fprintf(stderr, "------------------------------ before mult:\n");*/
	/*for (int i = 0; i < 5; ++i) {*/
	/*    for (int j = 0; j < 3; ++j) {*/
	/*        fprintf(stderr, "%5.1lf ", ((float *) Bs)[5 * j + i]);*/
	/*    }*/
	/*    fprintf(stderr, "\n");*/
	/*}*/
	/*fprintf(stderr, "------------------------------ B matrix\n");*/
	/*for (int i = 0; i < B.height; ++i) {*/
	/*    for (int j = 0; j < B.width; ++j) {*/
	/*        fprintf(stderr, "%5.1lf ", ((float *) B.els)[B.height * j + i]);*/
	/*    }*/
	/*    fprintf(stderr, "\n");*/
	/*}*/
	/*fprintf(stderr, "------------------------------\n");*/

	hipMemcpy(d_A.els, As, 5*5*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B.els, Bs, 5*3*sizeof(float), hipMemcpyHostToDevice);
	/*hipMemcpy(d_B.els, B.els, B.width*B.height*sizeof(float), hipMemcpyHostToDevice);*/

#endif

	hipblasStrsm('l',		/* side: a is on the left side of B (and this X) */
				'u',		/* uplo: upper triangular */
				'n',		/* transa: don't transpose */
				'n',		/* diag: NOT unit diagonal */
				5,			/* m: number of rows in B, and since 'l', it's also
							   the order of A */
				3,			/* n: number of columns in B */
				1.0,		/* alpha: alpha scalar */
				d_A.els,	/* a: 'A' matrix */
				5,			/* lda -- ??? */
				d_B.els,	/* b: 'B' matrix */
				5			/* ldb -- ??? */);

	hipMemcpy(Bs, d_B.els, 5*3*sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < 5; ++i) {
		for (int j = 0; j < 3; ++j) {
			fprintf(stderr, "%5.1lf ", ((float *) Bs)[5 * j + i]);
		}
		fprintf(stderr, "\n");
	}

	int e = cublasGetError();
	if (e == HIPBLAS_STATUS_NOT_INITIALIZED) {
		fprintf(stderr, "HIPBLAS_STATUS_NOT_INITIALIZED\n");
	} else if (e == HIPBLAS_STATUS_INVALID_VALUE) { 
		fprintf(stderr, "HIPBLAS_STATUS_INVALID_VALUE\n");
	} else if (e == HIPBLAS_STATUS_EXECUTION_FAILED) {
		fprintf(stderr, "HIPBLAS_STATUS_EXECUTION_FAILED\n");
	}

	return 0;
}
