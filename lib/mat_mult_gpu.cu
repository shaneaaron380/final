#include "hip/hip_runtime.h"
#include "mat_mult_gpu.h"

__global__ void MatMultKernel(const Matrix A, const Matrix B, Matrix C)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;
  int n = A.width;
  float S;

  for (int j = 0; j < n; j++) {
    S = B.els[i*n+j]; //S = B[i][j];
    for (int k = 0; k < i; k++) {
      S -= A.els[i*n+k] * C.els[k*n+j]; //S -= A[i][k] * C[k][j];
    }
    C.els[i*n+j] = S; //C[i][j] = S;
  }

}

// matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMultGPU(const Matrix A, const Matrix B, Matrix C)
{
	Matrix d_A, d_B, d_C;

	d_A.width = d_A.stride = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc((void**)&d_A.els, size);
	hipMemcpy(d_A.els, A.els, size, hipMemcpyHostToDevice);

	d_B.width = d_B.stride = B.width;
	d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc((void**)&d_B.els, size);
	hipMemcpy(d_B.els, B.els, size, hipMemcpyHostToDevice);

	d_C.width = d_C.stride = C.width;
	d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	hipMalloc((void**)&d_C.els, size);

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
	MatMultKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

	hipMemcpy(C.els, d_C.els, size, hipMemcpyDeviceToHost);

	hipFree(d_A.els);
	hipFree(d_B.els);
	hipFree(d_C.els);
}


