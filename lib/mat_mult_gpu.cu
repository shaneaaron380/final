#include "hip/hip_runtime.h"

#include "mat_mult_gpu.h"
#include "sys/time.h"
//#include "hip/hip_runtime.h"
#include "cuPrintf.cu"

#define SMEM_CACHE_SZ 32 
//#define THREADS_PER_BLOCK 512 

__global__ void MatMultKernel(const Matrix A, const Matrix B, Matrix C, const float alpha, const int N)
{
  int l = 0;
  //int j = (gridDim.x-1)*512 + threadIdx.x;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;
  //int ixn = 0;
  float S;

  if (j < N) {
    //cuPrintf("%d,%d : %d,%d : %d,%d\n", blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, threadIdx.x, threadIdx.y);
    //if ( (j % 20) == 0) cuPrintf("%d,%d,%d\n", j, gridDim.x, threadIdx.x);
    //for (int i = 0; i < N*N; i+=N) {
    for (int i = 0; i < N; i++) {
			//ixn = i * N;
			S = alpha*B.els[i*N+j]; //S = B[i][j];
      //cuPrintf("i=%d,j=%d, S=%f\n", i, j, S);
      for (int k = 0; k < i; k++) {
        //S -= A.els[i*n+k] * C.els[k*n+j]; //S -= A[i][k] * C[k][j];
        S -= A.els[l++] * C.els[k*N+j]; //S -= A[i][k] * C[k][j];
        //cuPrintf("i=%d,j=%d,k=%d, S=%f, A=%f, C=%f\n", i, j, k, S, A.els[i*n+k], C.els[k*n+j]);
      }
      C.els[i*N+j] = S; //C[i][j] = S;
    }
  }
}

__global__ void MatMultKernelShared(const Matrix A, const Matrix B, Matrix C, const float alpha, const int N)
{
  int l = 0, k = 0;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
	int t_idx = threadIdx.x;
	float S;

  __shared__ float As[SMEM_CACHE_SZ];

  //if (j == 0) cuPrintf("N=%d\n", N);
  //if (t_idx == 241) cuPrintf("j=%d, t_idx=%d\n", j, t_idx);
  //Init cache to zero
  //if (t_idx < SMEM_CACHE_SZ) As[t_idx] = 0;
 
  if (j < N) {
		//cuPrintf("j = %d, t_idx = %d\n", j, t_idx);
    //cuPrintf("%d,%d : %d,%d : %d,%d\n", blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, threadIdx.x, threadIdx.y);
   	C.els[j] = alpha*B.els[j];
    //if (t_idx == 241) cuPrintf("j=%d, C.els[j]=%f, B.els[j]=%f\n", j, C.els[j], B.els[j]);
		for (int i = 1; i < N; i++) {
      
			S = alpha*B.els[i*N+j]; //S = B[i][j];
      //if (t_idx == 0) cuPrintf("i=%d, j=%d, Sinit=%f\n", i, j, S);
			k = 0;
			while ((k + 10) <= i) {
        __syncthreads();
				if (t_idx < 10) As[t_idx] = A.els[l+t_idx];
      	__syncthreads();
        //if (t_idx == 0) cuPrintf("4: As[%d] = A.els[%d] = %f\n", t_idx, l+t_idx, As[t_idx]);
        
        //cuPrintf("A+1=%f, C+1=%f\n", As[k+1], C.els[(k+1)*N+j]);
				S -=  (As[0]  * C.els[k*N+j]) + \
							(As[1]  * C.els[(k+1)*N+j]) + \
							(As[2]  * C.els[(k+2)*N+j]) + \
							(As[3]  * C.els[(k+3)*N+j]) + \
							(As[4]  * C.els[(k+4)*N+j]) + \
							(As[5]  * C.els[(k+5)*N+j]) + \
							(As[6]  * C.els[(k+6)*N+j]) + \
							(As[7]  * C.els[(k+7)*N+j]) + \
							(As[8]  * C.els[(k+8)*N+j]) + \
							(As[9]  * C.els[(k+9)*N+j]);
        
        //if(t_idx == 0) cuPrintf("10: i=%d,j=%d,k=%d,As[0]=%f,A.els[l]=%f,C.els[%d]=%f,S=%f\n", i, j, k, As[0], A.els[l+t_idx], k*N+j, C.els[k*N+j], S);
        k+=10;
     	  l+=10;
      }
			while ((k + 4) <= i) {
        __syncthreads();
				if (t_idx < 4) As[t_idx] = A.els[l+t_idx];
      	__syncthreads();
        //if (t_idx == 0) cuPrintf("4: As[%d] = A.els[%d] = %f\n", t_idx, l+t_idx, As[t_idx]);
        
        //cuPrintf("A+1=%f, C+1=%f\n", As[k+1], C.els[(k+1)*N+j]);
				S -=  (As[0]  * C.els[k*N+j]) + \
							(As[1]  * C.els[(k+1)*N+j]) + \
							(As[2]  * C.els[(k+2)*N+j]) + \
							(As[3]  * C.els[(k+3)*N+j]); 
        
        //if(t_idx == 0) cuPrintf("4: i=%d,j=%d,k=%d,As[0]=%f,A.els[l]=%f,C.els[%d]=%f,S=%f\n", i, j, k, As[0], A.els[l+t_idx], k*N+j, C.els[k*N+j], S);
        k+=4;
     	  l+=4;
      }
      while (k < i)  {
        __syncthreads();
      	if (t_idx == 0) As[0] = A.els[l];
        __syncthreads();
        S -= As[0] * C.els[k*N+j]; //S -= A[i][k] * C[k][j];
        //if(t_idx == 0) cuPrintf("1: i=%d,j=%d,k=%d,As[0]=%f,A.els[l]=%f,C.els[%d]=%f,S=%f\n", i, j, k, As[0], A.els[l], k*N+j, C.els[k*N+j], S);
     	  k++;
     	  l++;
      }
      C.els[i*N+j] = S; //C[i][j] = S;
      //if ((i*N+j) == 241) cuPrintf("i=%d, j=%d, C.els[%d] = %f\n", i, j, i*N+j, C.els[i*N+j]);
    }
  }
}

void MatMultGPU(const Matrix A, const Matrix B, Matrix C, const float alpha)
{
	Matrix d_A, d_B, d_C;
  const int n = A.width;
	hipError_t cudaMallocReturnStatus;
	struct timeval timerValues;
	double start_time, end_time;
	double before_kernel, after_kernel;
	timerclear(&timerValues);	

  cudaPrintfInit();

	d_A.width = d_A.stride = A.width;
	d_A.height = A.height;
	size_t asize = ((A.width * A.height - A.width)/2) * sizeof(float);
	hipMalloc((void**)&d_A.els, asize);
	cudaMallocReturnStatus = hipMalloc((void**)&d_A.els, asize);
	if (cudaMallocReturnStatus == hipErrorOutOfMemory) {
		printf("ERROR: Couldn't allocate Matrix A on GPU, exiting\n"); exit(0);
	}
  TruncateMatrix(A);

	d_B.width = d_B.stride = B.width;
	d_B.height = B.height;
	size_t size = B.width * B.height * sizeof(float);
	hipMalloc((void**)&d_B.els, size);
	if (cudaMallocReturnStatus == hipErrorOutOfMemory) {
		printf("ERROR: Couldn't allocate Matrix B on GPU, exiting\n"); exit(0);
	}

	d_C.width = d_C.stride = C.width;
	d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	hipMalloc((void**)&d_C.els, size);
	if (cudaMallocReturnStatus == hipErrorOutOfMemory) {
		printf("ERROR: Couldn't allocate Matrix C on GPU, exiting\n"); exit(0);
	}

  int threadsPerBlock = n > 512 ? 512 : n;
  int blocksPerGrid = (n+threadsPerBlock-1)/threadsPerBlock;
  printf("grids=%d, threads=%d\n", blocksPerGrid, threadsPerBlock);
	
	//Get start time
	if (gettimeofday(&timerValues, NULL))
		printf("WARNING: Counldn't get start time of day\n");
	
	start_time = (double) timerValues.tv_sec	+ (double) (timerValues.tv_usec)/1000000;
	
	hipMemcpy(d_A.els, A.els, asize, hipMemcpyHostToDevice);
	hipMemcpy(d_B.els, B.els, size, hipMemcpyHostToDevice);
	
	if (gettimeofday(&timerValues, NULL))
		printf("WARNING: Counldn't get before kernel time of day\n");
	
	before_kernel = (double) timerValues.tv_sec	+ (double) (timerValues.tv_usec)/1000000;
	
	//MatMultKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, alpha, n);
	//Static shared memory
	MatMultKernelShared<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, alpha, n);
	//Dynamic shared memory
	//MatMultKernelShared<<<blocksPerGrid, threadsPerBlock, sizeof(float)*(n-1)>>>(d_A, d_B, d_C, alpha, n);
	hipDeviceSynchronize();	
	
	if (gettimeofday(&timerValues, NULL))
		printf("WARNING: Counldn't get after kernel time of day\n");
	
	after_kernel = (double) timerValues.tv_sec	+ (double) (timerValues.tv_usec)/1000000;
	
	hipMemcpy(C.els, d_C.els, size, hipMemcpyDeviceToHost);
  
	//Get end time
	if (gettimeofday(&timerValues, NULL))
		printf("WARNING: Counldn't get end time of day\n");
	
	end_time = (double) timerValues.tv_sec	+ (double) (timerValues.tv_usec)/1000000;
	//printf("End secs: %ld, End usecs: %ld, Total Time: %f\n", timerValues.tv_sec, timerValues.tv_usec, end_time-start_time);
	printf("Total Time: %f\n", end_time-start_time);
	printf("Kernel Time: %f\n", after_kernel-before_kernel);
	printf("Transfer Time: %f\n", (end_time-after_kernel)+(before_kernel-start_time));
  
  cudaPrintfDisplay(stdout,true);
  cudaPrintfEnd();

	hipFree(d_A.els);
	hipFree(d_B.els);
	hipFree(d_C.els);
}


