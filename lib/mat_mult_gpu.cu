#include "hip/hip_runtime.h"

#include "mat_mult_gpu.h"
#include "cuPrintf.cu"
#include "hip/hip_runtime.h"

__global__ void MatMultKernel(const Matrix A, const Matrix B, Matrix C, int n)
{
  int l = 0;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;
  float S;
 
  if (j < n) {
    //cuPrintf("%d,%d : %d,%d : %d,%d\n", blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, threadIdx.x, threadIdx.y);
    for (int i = 0; i < n; i++) {
      S = B.els[i*n+j]; //S = B[i][j];
      //cuPrintf("i=%d,j=%d, S=%f\n", i, j, S);
      for (int k = 0; k < i; k++) {
        //S -= A.els[i*n+k] * C.els[k*n+j]; //S -= A[i][k] * C[k][j];
        S -= A.els[l] * C.els[k*n+j]; //S -= A[i][k] * C[k][j];
        l++;
        //cuPrintf("i=%d,j=%d,k=%d, S=%f, A=%f, C=%f\n", i, j, k, S, A.els[i*n+k], C.els[k*n+j]);
      }
      C.els[i*n+j] = S; //C[i][j] = S;
    }
  }

}

void TruncateMatrix(Matrix A) {

  int k = 0;
  int n = A.width;
  //int size = (n*n-n)/2;

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      if (i == j) continue;
      if (j < i) {
        //assert(k<size);
        A.els[k] = A.els[i*n+j];
        k++;
      }
    }
  }
}

// matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMultGPU(const Matrix A, const Matrix B, Matrix C)
{
	Matrix d_A, d_B, d_C;

  // Initialize 
  hipInit(0); 
  
  // Get number of devices supporting CUDA 
  int deviceCount = 0; 
  hipGetDeviceCount(&deviceCount); 
  if (deviceCount == 0) { 
    printf("There is no device supporting CUDA.\n"); exit (0); 
  }
  printf("deviceCount=%d\n", deviceCount);
  int n = A.width;
	hipError_t cudaMallocReturnStatus;

  cudaPrintfInit();

	d_A.width = d_A.stride = A.width;
	d_A.height = A.height;
	//size_t size = A.width * A.height * sizeof(float);
	size_t size = ((A.width * A.height - A.width)/2) * sizeof(float);
	hipMalloc((void**)&d_A.els, size);
	cudaMallocReturnStatus = hipMalloc((void**)&d_A.els, size);
	if (cudaMallocReturnStatus == hipErrorOutOfMemory) {
		printf("Couldn't allocate Matrix A on GPU, exiting\n"); exit(0);
	}
  TruncateMatrix(A);
	hipMemcpy(d_A.els, A.els, size, hipMemcpyHostToDevice);

	d_B.width = d_B.stride = B.width;
	d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc((void**)&d_B.els, size);
	if (cudaMallocReturnStatus == hipErrorOutOfMemory) {
		printf("Couldn't allocate Matrix B on GPU, exiting\n"); exit(0);
	}
	hipMemcpy(d_B.els, B.els, size, hipMemcpyHostToDevice);

	d_C.width = d_C.stride = C.width;
	d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	hipMalloc((void**)&d_C.els, size);
	if (cudaMallocReturnStatus == hipErrorOutOfMemory) {
		printf("Couldn't allocate Matrix C on GPU, exiting\n"); exit(0);
	}

	//dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	//dim3 dimBlock(A.width, A.width);
	//dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
  //printf("dimBlock.x = %d, dimBlock.y = %d\n", dimBlock.x, dimBlock.y);
  int threadsPerBlock = 512;
  int blocksPerGrid = (n+threadsPerBlock-1)/threadsPerBlock;
  printf("grids=%d, threads=%d\n", blocksPerGrid, threadsPerBlock);
	//MatMultKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, A.width);
	MatMultKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);
  
  cudaPrintfDisplay(stdout,true);
  cudaPrintfEnd();

	hipMemcpy(C.els, d_C.els, size, hipMemcpyDeviceToHost);

	hipFree(d_A.els);
	hipFree(d_B.els);
	hipFree(d_C.els);
}


