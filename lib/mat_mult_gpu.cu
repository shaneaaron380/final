#include "hip/hip_runtime.h"

#include "mat_mult_gpu.h"
#include "sys/time.h"
//#include "hip/hip_runtime.h"
#include "cuPrintf.cu"

__global__ void MatMultKernel(const Matrix A, const Matrix B, Matrix C, const float alpha, int n)
{
  int l = 0;
  //int j = (gridDim.x-1)*512 + threadIdx.x;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;
  float S;

  if (j < n) {
    //cuPrintf("%d,%d : %d,%d : %d,%d\n", blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, threadIdx.x, threadIdx.y);
    //if ( (j % 20) == 0) cuPrintf("%d,%d,%d\n", j, gridDim.x, threadIdx.x);
    for (int i = 0; i < n; i++) {
      S = alpha*B.els[i*n+j]; //S = B[i][j];
      //cuPrintf("i=%d,j=%d, S=%f\n", i, j, S);
      for (int k = 0; k < i; k++) {
        //S -= A.els[i*n+k] * C.els[k*n+j]; //S -= A[i][k] * C[k][j];
        S -= A.els[l] * C.els[k*n+j]; //S -= A[i][k] * C[k][j];
        l++;
        //cuPrintf("i=%d,j=%d,k=%d, S=%f, A=%f, C=%f\n", i, j, k, S, A.els[i*n+k], C.els[k*n+j]);
      }
      C.els[i*n+j] = S; //C[i][j] = S;
    }
  }
}


__global__ void MatMultKernelShared(const Matrix A, const Matrix B, Matrix C, const float alpha, const int n)
{
  int l = 0;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;
  int m = n%512 ? n%512+1 : n%512;
	float S;

  extern __shared__ float As[];
 
  if (j < n) {
    //cuPrintf("%d,%d : %d,%d : %d,%d\n", blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, threadIdx.x, threadIdx.y);
    for (int i = 0; i < n; i++) {
      
      __syncthreads();
			for (int o = 0; o < m; o++) {
      	int p = o*512+threadIdx.x;
				if (p < i) { 
        	As[p] = A.els[l+p];
        	//cuPrintf("j=%d, i=%d,l+j=%d, As[l+j]=%f\n", j, i,l+j, As[j]);
      	}
				else break;
			}
      __syncthreads();
      
      S = alpha*B.els[i*n+j]; //S = B[i][j];
      //cuPrintf("i=%d,j=%d, S=%f\n", i, j, S);
      for (int k = 0; k < i; k++) {
        //S -= A.els[i*n+k] * C.els[k*n+j]; //S -= A[i][k] * C[k][j];
        //S -= A.els[l] * C.els[k*n+j]; //S -= A[i][k] * C[k][j];
        S -= As[k] * C.els[k*n+j]; //S -= A[i][k] * C[k][j];
        //cuPrintf("i=%d,j=%d,k=%d, S=%f, A=%f, C=%f\n", i, j, k, S, As[k], C.els[k*n+j]);
      }
      l += i;
      C.els[i*n+j] = S; //C[i][j] = S;
    }
  }

}

// this is now in matrix.h since it's shared
//void TruncateMatrix(Matrix A) {
//
//  int k = 0;
//  int n = A.width;
//  //int size = (n*n-n)/2;
//
//  for (int i = 0; i < n; i++) {
//    for (int j = 0; j < n; j++) {
//      if (i == j) continue;
//      if (j < i) {
//        //assert(k<size);
//        A.els[k] = A.els[i*n+j];
//        k++;
//      }
//    }
//  }
//}

// matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMultGPU(const Matrix A, const Matrix B, Matrix C, const float alpha)
{
	Matrix d_A, d_B, d_C;

  // Initialize 
  //hipInit(0); 
  
  // Get number of devices supporting CUDA 
  //int deviceCount = 0; 
  //hipGetDeviceCount(&deviceCount); 
  //if (deviceCount == 0) { 
  //  printf("There is no device supporting CUDA.\n"); exit (0); 
  //}
  //printf("deviceCount=%d\n", deviceCount);
  const int n = A.width;
	hipError_t cudaMallocReturnStatus;
	struct timeval timerValues;
	double start_time, end_time;
	timerclear(&timerValues);	

  cudaPrintfInit();

	d_A.width = d_A.stride = A.width;
	d_A.height = A.height;
	//size_t size = A.width * A.height * sizeof(float);
	size_t asize = ((A.width * A.height - A.width)/2) * sizeof(float);
	hipMalloc((void**)&d_A.els, asize);
	cudaMallocReturnStatus = hipMalloc((void**)&d_A.els, asize);
	if (cudaMallocReturnStatus == hipErrorOutOfMemory) {
		printf("ERROR: Couldn't allocate Matrix A on GPU, exiting\n"); exit(0);
	}
  TruncateMatrix(A);
  //for (int i = 0; i < 3; i++) printf("A[%d] = %f\n", i, A.els[i]);
	//hipMemcpy(d_A.els, A.els, size, hipMemcpyHostToDevice);

	d_B.width = d_B.stride = B.width;
	d_B.height = B.height;
	size_t size = B.width * B.height * sizeof(float);
	hipMalloc((void**)&d_B.els, size);
	if (cudaMallocReturnStatus == hipErrorOutOfMemory) {
		printf("ERROR: Couldn't allocate Matrix B on GPU, exiting\n"); exit(0);
	}
	//hipMemcpy(d_B.els, B.els, size, hipMemcpyHostToDevice);

	d_C.width = d_C.stride = C.width;
	d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	hipMalloc((void**)&d_C.els, size);
	if (cudaMallocReturnStatus == hipErrorOutOfMemory) {
		printf("ERROR: Couldn't allocate Matrix C on GPU, exiting\n"); exit(0);
	}

  int threadsPerBlock = 512;
  int blocksPerGrid = (n+threadsPerBlock-1)/threadsPerBlock;
	
	//Get start time
	if (gettimeofday(&timerValues, NULL))
		printf("WARNING: Counldn't get start time of day\n");
	
	//if (timerisset(&timerValues)) 
	start_time = (double) timerValues.tv_sec	+ (double) (timerValues.tv_usec)/1000000;
	//printf("Start secs: %ld, Start usecs: %ld, Time: %f\n", timerValues.tv_sec, timerValues.tv_usec, start_time);
	
	hipMemcpy(d_A.els, A.els, asize, hipMemcpyHostToDevice);
	hipMemcpy(d_B.els, B.els, size, hipMemcpyHostToDevice);

  //printf("grids=%d, threads=%d\n", blocksPerGrid, threadsPerBlock);
	MatMultKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, alpha, n);
	//MatMultKernelShared<<<blocksPerGrid, threadsPerBlock, sizeof(float)*(n-1)>>>(d_A, d_B, d_C, alpha, n);
	//MatMultKernel<<<1, 3, sizeof(float)*(n-1)>>>(d_A, d_B, d_C, alpha, n);
  
	hipMemcpy(C.els, d_C.els, size, hipMemcpyDeviceToHost);
	//Get end time
	if (gettimeofday(&timerValues, NULL))
		printf("WARNING: Counldn't get end time of day\n");
	
	//if (timerisset(&timerValues)) 
	end_time = (double) timerValues.tv_sec	+ (double) (timerValues.tv_usec)/1000000;
	//printf("End secs: %ld, End usecs: %ld, Total Time: %f\n", timerValues.tv_sec, timerValues.tv_usec, end_time-start_time);
	printf("Total Time: %f\n", end_time-start_time);
  
  cudaPrintfDisplay(stdout,true);
  cudaPrintfEnd();

	hipFree(d_A.els);
	hipFree(d_B.els);
	hipFree(d_C.els);
}


