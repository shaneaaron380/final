#include "hip/hip_runtime.h"

#include "mat_mult_gpu.h"
#include "cuPrintf.cu"
#include "sys/time.h"
//#include "hip/hip_runtime.h"

__global__ void MatMultKernel(const Matrix A, const Matrix B, Matrix C, const float alpha, int n)
{
  int l = 0;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;
  float S;
 
  if (j < n) {
    //cuPrintf("%d,%d : %d,%d : %d,%d\n", blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, threadIdx.x, threadIdx.y);
    for (int i = 0; i < n; i++) {
      S = alpha*B.els[i*n+j]; //S = B[i][j];
      //cuPrintf("i=%d,j=%d, S=%f\n", i, j, S);
      for (int k = 0; k < i; k++) {
        //S -= A.els[i*n+k] * C.els[k*n+j]; //S -= A[i][k] * C[k][j];
        S -= A.els[l] * C.els[k*n+j]; //S -= A[i][k] * C[k][j];
        l++;
        //cuPrintf("i=%d,j=%d,k=%d, S=%f, A=%f, C=%f\n", i, j, k, S, A.els[i*n+k], C.els[k*n+j]);
      }
      C.els[i*n+j] = S; //C[i][j] = S;
    }
  }

}

void TruncateMatrix(Matrix A) {

  int k = 0;
  int n = A.width;
  //int size = (n*n-n)/2;

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      if (i == j) continue;
      if (j < i) {
        //assert(k<size);
        A.els[k] = A.els[i*n+j];
        k++;
      }
    }
  }
}

// matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMultGPU(const Matrix A, const Matrix B, Matrix C, const float alpha)
{
	Matrix d_A, d_B, d_C;

  // Initialize 
  //hipInit(0); 
  
  // Get number of devices supporting CUDA 
  //int deviceCount = 0; 
  //hipGetDeviceCount(&deviceCount); 
  //if (deviceCount == 0) { 
  //  printf("There is no device supporting CUDA.\n"); exit (0); 
  //}
  //printf("deviceCount=%d\n", deviceCount);
  int n = A.width;
	hipError_t cudaMallocReturnStatus;
	struct timeval timerValues;
	double start_time, end_time;
	timerclear(&timerValues);	

  cudaPrintfInit();

	d_A.width = d_A.stride = A.width;
	d_A.height = A.height;
	//size_t size = A.width * A.height * sizeof(float);
	size_t size = ((A.width * A.height - A.width)/2) * sizeof(float);
	hipMalloc((void**)&d_A.els, size);
	cudaMallocReturnStatus = hipMalloc((void**)&d_A.els, size);
	if (cudaMallocReturnStatus == hipErrorOutOfMemory) {
		printf("ERROR: Couldn't allocate Matrix A on GPU, exiting\n"); exit(0);
	}
  TruncateMatrix(A);
	//hipMemcpy(d_A.els, A.els, size, hipMemcpyHostToDevice);

	d_B.width = d_B.stride = B.width;
	d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc((void**)&d_B.els, size);
	if (cudaMallocReturnStatus == hipErrorOutOfMemory) {
		printf("ERROR: Couldn't allocate Matrix B on GPU, exiting\n"); exit(0);
	}
	//hipMemcpy(d_B.els, B.els, size, hipMemcpyHostToDevice);

	d_C.width = d_C.stride = C.width;
	d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	hipMalloc((void**)&d_C.els, size);
	if (cudaMallocReturnStatus == hipErrorOutOfMemory) {
		printf("ERROR: Couldn't allocate Matrix C on GPU, exiting\n"); exit(0);
	}

	
	//Get start time
	if (gettimeofday(&timerValues, NULL))
		printf("WARNING: Counldn't get start time of day\n");
	
	//if (timerisset(&timerValues)) 
	start_time = (double) timerValues.tv_sec	+ (double) (timerValues.tv_usec)/1000000;
	printf("Start secs: %ld, Start usecs: %ld, Time: %f\n", timerValues.tv_sec, timerValues.tv_usec, start_time);
	
	hipMemcpy(d_A.els, A.els, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B.els, B.els, size, hipMemcpyHostToDevice);

	//dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	//dim3 dimBlock(A.width, A.width);
	//dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
  //printf("dimBlock.x = %d, dimBlock.y = %d\n", dimBlock.x, dimBlock.y);
  int threadsPerBlock = 512;
  int blocksPerGrid = (n+threadsPerBlock-1)/threadsPerBlock;
  printf("grids=%d, threads=%d\n", blocksPerGrid, threadsPerBlock);
	//MatMultKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, A.width);
	MatMultKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, alpha, n);
  
  cudaPrintfDisplay(stdout,true);
  cudaPrintfEnd();

	hipMemcpy(C.els, d_C.els, size, hipMemcpyDeviceToHost);
	//Get end time
	if (gettimeofday(&timerValues, NULL))
		printf("WARNING: Counldn't get end time of day\n");
	
	//if (timerisset(&timerValues)) 
	end_time = (double) timerValues.tv_sec	+ (double) (timerValues.tv_usec)/1000000;
	printf("End secs: %ld, End usecs: %ld, Total Time: %f\n", timerValues.tv_sec, timerValues.tv_usec, end_time-start_time);

	hipFree(d_A.els);
	hipFree(d_B.els);
	hipFree(d_C.els);
}


